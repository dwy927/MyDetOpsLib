#include "pytorch_cuda_helper.hpp"
#include "nms_cuda_kernel.cuh"

Tensor NMSCUDAKernelLauncher(Tensor boxes, Tensor scores, float iou_threshold, int offset) {
  at::cuda::CUDAGuard device_guard(boxes.device());
  if (boxes.numel() == 0) {
    return at::empty({0}, boxes.options().dtype(at::kLong));
  }
  auto order_t = std::get<1>(scores.sort(0, true));
  auto boxes_sorted = boxes.index_select(0, order_t);
  int boxes_num = boxes.size(0);
  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  Tensor mask =
    at::empty({boxes_num, col_blocks}, boxes.options().dtype(at::kLong));
  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  nms_cuda<<<blocks, threads, 0, stream>>>(
      boxes_num, iou_threshold, offset, boxes_sorted.data_ptr<float>(),
      (unsigned long long*)mask.data_ptr<int64_t>());

  at::Tensor mask_cpu = mask.to(at::kCPU);
  unsigned long long* mask_host =
      (unsigned long long*)mask_cpu.data_ptr<int64_t>();

  std::vector<unsigned long long> remv(col_blocks, 0);
  // memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep_t = at::zeros(
      {boxes_num}, boxes.options().dtype(at::kBool).device(at::kCPU));
  bool* keep = keep_t.data_ptr<bool>();

  for (int i=0; i < boxes_num; ++i) {
    const int nblock = i / threadsPerBlock;
    const int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep[i] = true;
      unsigned long long* p = mask_host + i * col_blocks;
      for (int j = nblock; j < col_blocks; ++j) {
        remv[j] |= p[j];
      }
    }
  }
  AT_CUDA_CHECK(hipGetLastError());
  return order_t.masked_select(keep_t.to(at::kCUDA));
}
