#include "pytorch_cuda_helper.hpp"
#include "roi_pool_cuda_kernel.cuh"

void RoIPoolForwardCUDAKernelLauncher(Tensor input, Tensor rois, Tensor output,
                                      Tensor argmax, int pooled_height,
                                      int pooled_width, float spatial_scale) {

  const int output_size = output.numel();
  const int channels = input.size(1);
  const int height = input.size(2);
  const int width = input.size(3);

  at::cuda::CUDAGuard device_guard(input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    input.scalar_type(), "roi_pool_forward_cuda_kernel", [&] {
      roi_pool_forward_cuda_kernel<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
            output_size, input.data_ptr<scalar_t>(),
            rois.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
            argmax.data_ptr<int>(), pooled_height, pooled_width,
            static_cast<scalar_t>(spatial_scale), channels, height, width);
    });

  AT_CUDA_CHECK(hipGetLastError());
}

void RoIPoolBackwardCUDAKernelLauncher(Tensor grad_output, Tensor rois,
                                       Tensor argmax, Tensor grad_input,
                                       int pooled_height, int pooled_width) {
  const int output_size = grad_output.numel();
  const int channels = grad_input.size(1);
  const int height = grad_input.size(2);
  const int width = grad_input.size(3);

  at::cuda::CUDAGuard device_guard(grad_output.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_output.scalar_type(), "roi_pool_backward_cuda_kernel", [&] {
      roi_pool_backward_cuda_kernel<scalar_t>
        <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
            output_size, grad_output.data_ptr<scalar_t>(),
            rois.data_ptr<scalar_t>(), argmax.data_ptr<int>(),
            grad_input.data_ptr<scalar_t>(), pooled_height, pooled_width,
            channels, height, width);
    });

  AT_CUDA_CHECK(hipGetLastError());
}
